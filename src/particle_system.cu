#include <iostream>
#include <hip/hip_runtime.h>
#include <math.h>
#include <chrono>
#include <cstdlib>
#include <ctime>

struct Particle {
    float x, y;
    float vx, vy;
    float r, g, b;
};

// 1. Основная функция обновления частиц
__global__ void updateParticles(Particle* particles, int numParticles, float dt) {
    int idx = threadIdx.x + blockIdx.x * blockDim.x;
    if (idx >= numParticles) return;

    // Применение гравитации
    particles[idx].vy -= 9.8f * dt;
    
    // Обновление позиции
    particles[idx].x += particles[idx].vx * dt;
    particles[idx].y += particles[idx].vy * dt;
    
    // Отскок от земли
    if (particles[idx].y < 0) {
        particles[idx].y = 0;
        particles[idx].vy = -particles[idx].vy * 0.8f;
    }
    
    // Отскок от стен
    if (particles[idx].x < -10 || particles[idx].x > 10) {
        particles[idx].vx = -particles[idx].vx * 0.8f;
        particles[idx].x = (particles[idx].x < -10) ? -10 : 10;
    }
}

// 2. Улучшенная функция обработки столкновений
__global__ void handleParticleCollisions(Particle* particles, int numParticles) {
    int idx = threadIdx.x + blockIdx.x * blockDim.x;
    if (idx >= numParticles) return;

    float radius = 0.3f;
    float restitution = 0.9f; // Коэффициент восстановления
    
    // Проверяем только близлежащие частицы (для оптимизации производительности)
    for (int j = idx + 1; j < min(idx + 50, numParticles); j++) {
        float dx = particles[idx].x - particles[j].x;
        float dy = particles[idx].y - particles[j].y;
        float distance = sqrtf(dx*dx + dy*dy);
        
        if (distance < radius * 2 && distance > 0.001f) {
            // Единичный нормальный вектор
            float nx = dx / distance;
            float ny = dy / distance;
            
            // Относительная скорость
            float dvx = particles[idx].vx - particles[j].vx;
            float dvy = particles[idx].vy - particles[j].vy;
            float velocity_along_normal = dvx * nx + dvy * ny;
            
            // Не сталкиваемся, если частицы удаляются друг от друга
            if (velocity_along_normal > 0) continue;
            
            // Импульс столкновения
            float impulse = -(1.0f + restitution) * velocity_along_normal;
            impulse /= 2.0f;
            
            // Применение импульса
            particles[idx].vx += impulse * nx;
            particles[idx].vy += impulse * ny;
            particles[j].vx -= impulse * nx;
            particles[j].vy -= impulse * ny;
            
            // Коррекция перекрытия
            float overlap = radius * 2 - distance;
            particles[idx].x += nx * overlap * 0.5f;
            particles[idx].y += ny * overlap * 0.5f;
            particles[j].x -= nx * overlap * 0.5f;
            particles[j].y -= ny * overlap * 0.5f;
        }
    }
}

// 3. Функция сопротивления воздуха (опционально)
__global__ void applyAirResistance(Particle* particles, int numParticles, float dt) {
    int idx = threadIdx.x + blockIdx.x * blockDim.x;
    if (idx >= numParticles) return;

    float airResistance = 0.99f; // Простое сопротивление воздуха
    
    particles[idx].vx *= airResistance;
    particles[idx].vy *= airResistance;
}

// 4. Вспомогательные функции для управления памятью
void copyToDevice(Particle* d_particles, Particle* h_particles, int numParticles) {
    hipMemcpy(d_particles, h_particles, numParticles * sizeof(Particle), hipMemcpyHostToDevice);
}

void copyToHost(Particle* h_particles, Particle* d_particles, int numParticles) {
    hipMemcpy(h_particles, d_particles, numParticles * sizeof(Particle), hipMemcpyDeviceToHost);
}

int main() {
    const int NUM_PARTICLES = 3000;
    const float DT = 0.016f;
    
    // Initialize random seed
    std::srand(static_cast<unsigned>(std::time(0)));
    
    auto start = std::chrono::high_resolution_clock::now();
    
    // 1. Создание частиц на CPU
    Particle* h_particles = new Particle[NUM_PARTICLES];
    for (int i = 0; i < NUM_PARTICLES; i++) {
        h_particles[i] = {
            (float)(rand() % 15 - 7),   // x: -7 to 7
            (float)(rand() % 8 + 2),    // y: 2 to 10
            (float)(rand() % 6 - 3),    // vx: -3 to 3
            (float)(rand() % 4 - 6),    // vy: -6 to -2
            (float)rand() / RAND_MAX,
            (float)rand() / RAND_MAX,  
            (float)rand() / RAND_MAX
        };
    }
    
    // 2. Выделение памяти на GPU
    Particle* d_particles;
    hipMalloc(&d_particles, NUM_PARTICLES * sizeof(Particle));
    
    // 3. Копирование данных на GPU
    copyToDevice(d_particles, h_particles, NUM_PARTICLES);
    
    std::cout << "Simulating " << NUM_PARTICLES << " particles with advanced physics...\n";
    
    // 4. Настройка блоков и потоков
    int blockSize = 256;
    int numBlocks = (NUM_PARTICLES + blockSize - 1) / blockSize;
    
    // 5. Основное моделирование
    for (int step = 0; step < 100; step++) {
        // Основное обновление движения
        updateParticles<<<numBlocks, blockSize>>>(d_particles, NUM_PARTICLES, DT);
        hipDeviceSynchronize();
        
        // Сопротивление воздуха (каждые 3 шага)
        if (step % 3 == 0) {
            applyAirResistance<<<numBlocks, blockSize>>>(d_particles, NUM_PARTICLES, DT);
            hipDeviceSynchronize();
        }
        
        // Столкновения между частицами (каждый шаг)
        handleParticleCollisions<<<numBlocks, blockSize>>>(d_particles, NUM_PARTICLES);
        hipDeviceSynchronize();
        
        if (step % 20 == 0) {
            std::cout << "Step " << step << " - Physics simulation running...\n";
        }
    }
    
    // 6. Получение результатов с GPU
    copyToHost(h_particles, d_particles, NUM_PARTICLES);
    
    auto end = std::chrono::high_resolution_clock::now();
    auto duration = std::chrono::duration_cast<std::chrono::milliseconds>(end - start);
    
    // 7. Вывод результатов
    std::cout << "\n=== SIMULATION RESULTS ===" << std::endl;
    std::cout << "Execution time: " << duration.count() << " ms" << std::endl;
    std::cout << "First particle position: " << h_particles[0].x << ", " << h_particles[0].y << std::endl;
    
    // Дополнительный анализ
    float avgX = 0, avgY = 0;
    for (int i = 0; i < NUM_PARTICLES; i++) {
        avgX += h_particles[i].x;
        avgY += h_particles[i].y;
    }
    avgX /= NUM_PARTICLES;
    avgY /= NUM_PARTICLES;
    
    std::cout << "Average position: " << avgX << ", " << avgY << std::endl;
    std::cout << "Physics simulation completed successfully!" << std::endl;
    
    // 8. Очистка
    delete[] h_particles;
    hipFree(d_particles);
    
    return 0;
}
